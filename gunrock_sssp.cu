#include <torch/extension.h>
#include <iostream>

#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>

#include <gunrock/applications/sssp/sssp_implementation.hxx>

namespace py = pybind11;

using namespace gunrock;
using namespace memory;

// --
// Helpers

template <typename T>
auto numpy2cuda(py::array_t<T> x) {
  py::buffer_info ha = x.request();
  T* x_hptr          = reinterpret_cast<T*>(ha.ptr);
  
  T* x_dptr;
  hipMalloc(&x_dptr, ha.shape[0] * sizeof(T));
  hipMemcpy(x_dptr, x_hptr, ha.shape[0] * sizeof(T), hipMemcpyHostToDevice);
  return x_dptr;
}

template <typename T>
void cuda2numpy(py::array_t<T> x, T* x_dptr) {
  py::buffer_info ha = x.request();
  T* x_hptr          = reinterpret_cast<T*>(ha.ptr);

  hipMemcpy(x_hptr, x_dptr, ha.shape[0] * sizeof(T), hipMemcpyDeviceToHost);
}

// --
// Runner

template<typename vertex_t, typename edge_t, typename weight_t>
void gunrock_sssp(
  vertex_t      n_vertices,
  edge_t        n_edges,
  torch::Tensor offsets_arr,
  torch::Tensor indices_arr,
  torch::Tensor data_arr,
  vertex_t      single_source,
  torch::Tensor distances_arr,
  torch::Tensor predecessors_arr
) {
  auto d_offsets      = offsets_arr.data_ptr<vertex_t>();
  auto d_indices      = indices_arr.data_ptr<edge_t>();
  auto d_data         = data_arr.data_ptr<weight_t>();
  auto d_distances    = distances_arr.data_ptr<weight_t>();
  auto d_predecessors = predecessors_arr.data_ptr<vertex_t>();
  
  // Build graph + meta
  auto G = graph::build::from_csr_t<memory_space_t::device>(
    n_vertices,
    n_vertices,
    n_edges,
    d_offsets,
    d_indices,
    d_data
  );

  auto meta = graph::build::meta_t<vertex_t, edge_t, weight_t>(
    n_vertices,
    n_vertices,
    n_edges
  );

  // Run
  float elapsed = sssp::run(
    G,
    meta,
    single_source,
    d_distances,
    d_predecessors
  );
}


PYBIND11_MODULE(gunrock_sssp, m) {
  m.def("gunrock_sssp", gunrock_sssp<int,int,float>);
}
