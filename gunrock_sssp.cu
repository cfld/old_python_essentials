#include <sstream>
#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>

#include <gunrock/applications/sssp/sssp_implementation.hxx>

namespace py = pybind11;

using namespace gunrock;
using namespace memory;

// --
// Helpers

template <typename T>
auto numpy2cuda(py::array_t<T> x) {
  py::buffer_info ha = x.request();
  T* x_hptr          = reinterpret_cast<T*>(ha.ptr);
  
  T* x_dptr;
  hipMalloc(&x_dptr, ha.shape[0] * sizeof(T));
  hipMemcpy(x_dptr, x_hptr, ha.shape[0] * sizeof(T), hipMemcpyHostToDevice);
  return x_dptr;
}

template <typename T>
void cuda2numpy(py::array_t<T> x, T* x_dptr) {
  py::buffer_info ha = x.request();
  T* x_hptr          = reinterpret_cast<T*>(ha.ptr);

  hipMemcpy(x_hptr, x_dptr, ha.shape[0] * sizeof(T), hipMemcpyDeviceToHost);
}

// --
// Runner

template<typename vertex_t, typename edge_t, typename weight_t>
void gunrock_sssp(
  vertex_t              n_vertices,
  edge_t                n_edges,
  py::array_t<vertex_t> offsets_arr,
  py::array_t<edge_t>   indices_arr,
  py::array_t<weight_t> data_arr,
  vertex_t              single_source,
  py::array_t<weight_t> distances_arr,
  py::array_t<vertex_t> predecessors_arr
) {
  
  // Copy data to GPU
  auto d_offsets      = numpy2cuda(offsets_arr);
  auto d_indices      = numpy2cuda(indices_arr);
  auto d_data         = numpy2cuda(data_arr);
  auto d_distances    = numpy2cuda(distances_arr);
  auto d_predecessors = numpy2cuda(predecessors_arr);
  
  // Build graph + meta
  auto G = graph::build::from_csr_t<memory_space_t::device>(
    n_vertices,
    n_vertices,
    n_edges,
    d_offsets,
    d_indices,
    d_data
  );

  auto meta = graph::build::meta_t<vertex_t, edge_t, weight_t>(
    n_vertices,
    n_vertices,
    n_edges
  );

  // Run
  float elapsed = sssp::run(
    G,
    meta,
    single_source,
    d_distances,
    d_predecessors
  );
  
  // Copy results back to numpy
  cuda2numpy(distances_arr, d_distances);
  cuda2numpy(predecessors_arr, d_predecessors);
  
  // Free memory
  hipFree(d_offsets);
  hipFree(d_indices);
  hipFree(d_data);
  hipFree(d_distances);
  hipFree(d_predecessors);
}

PYBIND11_MODULE(gunrock_sssp, m) {
  m.def("gunrock_sssp", gunrock_sssp<int,int,float>);
}
